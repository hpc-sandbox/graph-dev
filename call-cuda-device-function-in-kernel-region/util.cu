
#include <hip/hip_runtime.h>
#include <iostream>

__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}

template<typename T>
__device__ T fetch_and_add(T *x, T inc) 
{
    T orig_val = myAtomicAdd(x,inc);
    return orig_val;
}

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

//template __device__ long fetch_and_add<long>(long*, long);
//template __device__ float fetch_and_add<float>(float*, float);
//template __device__ double fetch_and_add<double>(double*, double);


template __device__ int fetch_and_add<int>(int*, int);
