
#include <hip/hip_runtime.h>
#include <iostream>

__device__ int myAtomicAdd(int* address, int incr)
{
  int old = *address, assumed;

  do 
  {
    assumed = old;
    old = atomicCAS(address, assumed, incr + assumed);
  } while (assumed != old);
  
  return old;
}

template<typename T>
__device__ T fetch_and_add(T *x, T inc) 
{
    T orig_val = myAtomicAdd(x,inc);
    return orig_val;
}

__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

//template __device__ long fetch_and_add<long>(long*, long);
//template __device__ float fetch_and_add<float>(float*, float);
//template __device__ double fetch_and_add<double>(double*, double);


template __device__ int fetch_and_add<int>(int*, int);
